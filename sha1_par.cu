#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "sha1.cuh"

#include <stdlib.h>
#include <memory.h>

/****************************** MACROS ******************************/
#define SHA1_BLOCK_SIZE 20              // SHA1 outputs a 20 byte digest

/**************************** DATA TYPES ****************************/
typedef struct {
	BYTE data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[5];
	WORD k[4];
} CUDA_SHA1_CTX;

/****************************** MACROS ******************************/
#ifndef ROTLEFT
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#endif

/*********************** FUNCTION DEFINITIONS ***********************/
__device__  __forceinline__ void cuda_sha1_transform(CUDA_SHA1_CTX *ctx, const BYTE data[])
{
	WORD a, b, c, d, e, i, j, t, m[80];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) + (data[j + 1] << 16) + (data[j + 2] << 8) + (data[j + 3]);
	for ( ; i < 80; ++i) {
		m[i] = (m[i - 3] ^ m[i - 8] ^ m[i - 14] ^ m[i - 16]);
		m[i] = (m[i] << 1) | (m[i] >> 31);
	}

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];

	for (i = 0; i < 20; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (~b & d)) + e + ctx->k[0] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for ( ; i < 40; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[1] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for ( ; i < 60; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (b & d) ^ (c & d))  + e + ctx->k[2] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for ( ; i < 80; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[3] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
}

__device__ void cuda_sha1_init(CUDA_SHA1_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xEFCDAB89;
	ctx->state[2] = 0x98BADCFE;
	ctx->state[3] = 0x10325476;
	ctx->state[4] = 0xc3d2e1f0;
	ctx->k[0] = 0x5a827999;
	ctx->k[1] = 0x6ed9eba1;
	ctx->k[2] = 0x8f1bbcdc;
	ctx->k[3] = 0xca62c1d6;
}

__device__ void cuda_sha1_update(CUDA_SHA1_CTX *ctx, const BYTE data[], size_t len)
{
	size_t i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			cuda_sha1_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void cuda_sha1_final(CUDA_SHA1_CTX *ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		cuda_sha1_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	cuda_sha1_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and MD uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
	}
}

__global__ void kernel_sha1_hash(BYTE* indata, WORD inlen, BYTE* outdata, WORD n_batch)
{
	WORD thread = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread >= n_batch)
	{
		return;
	}
	BYTE* in = indata  + thread * inlen;
	BYTE* out = outdata  + thread * SHA1_BLOCK_SIZE;
	CUDA_SHA1_CTX ctx;
	cuda_sha1_init(&ctx);
	cuda_sha1_update(&ctx, in, inlen);
	cuda_sha1_final(&ctx, out);
}

void mcm_cuda_sha1_hash_batch(BYTE* in, WORD inlen, BYTE* out, WORD n_batch)
{
	BYTE *cuda_indata;
	BYTE *cuda_outdata;
	hipMalloc(&cuda_indata, inlen * n_batch);
	hipMalloc(&cuda_outdata, SHA1_BLOCK_SIZE * n_batch);
	hipMemcpy(cuda_indata, in, inlen * n_batch, hipMemcpyHostToDevice);

	WORD thread = 256;
	WORD block = (n_batch + thread - 1) / thread;
	kernel_sha1_hash << < block, thread >> > (cuda_indata, inlen, cuda_outdata, n_batch);
	hipMemcpy(out, cuda_outdata, SHA1_BLOCK_SIZE * n_batch, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error cuda sha1 hash:%d %s \n", error, hipGetErrorString(error));
	}
	hipFree(cuda_indata);
	hipFree(cuda_outdata);
}

int get_hash(char *start, char *all, int depth)
{
    if(depth > 20) {
        return 0;
    }
    BYTE* cuda_sha1_start;
    WORD cuda_sha1_start_size;
    WORD cuda_sha1_res_size;
    BYTE* cuda_sha1_res;
    cuda_sha1_res_size = reinterpret_cast<WORD>((unsigned int)64);
    cuda_sha1_res = (BYTE*)malloc(64 * SHA1_BLOCK_SIZE * sizeof(BYTE));
    char buffer[40];
    char temp[40];
    char target[40];
    depth++;
    for(int i = 97; i < 123; i++) {
        memset(target, '\0', sizeof(target));
        memset(buffer, '\0', sizeof(buffer));
        memset(temp, '\0', sizeof(temp));
        sprintf(target, "%s%c", start, i);
        cuda_sha1_start = reinterpret_cast<BYTE*>(target);
        cuda_sha1_start_size = reinterpret_cast<WORD>((unsigned int)strlen(target));
        mcm_cuda_sha1_hash_batch(cuda_sha1_start, cuda_sha1_start_size, cuda_sha1_res, cuda_sha1_res_size);
        for(int z =0;z<20;z++) {
            sprintf(temp, "%02x", cuda_sha1_res[z]);
            strcat(buffer, temp);
        }
        printf("Comparing:%s %s %s %d\n", target, buffer, all, strcmp(buffer, all));
        if(strcmp(buffer, all) == 0 ){
            printf("Found target:%s\n", target);
            return 1;
        }
    }
    for(int i = 97; i < 123; i++) {
        sprintf(target, "%s%c", start, i);
        if(1 == get_hash(target, all, depth)) {
            return 1;
        }
    }
    return 0;
}

int main()
{
    char start[256], all[256];
    printf("Start:");
    fgets(start, sizeof(start), stdin);
    size_t ln = strlen(start)-1;
    if (start[ln] == '\n')
        start[ln] = '\0';
    printf("All:");
    fgets(all, sizeof(all), stdin);
    size_t ln_all = strlen(all)-1;
    if (all[ln_all] == '\n')
        all[ln_all] = '\0';
    get_hash(start, all, 0);
}